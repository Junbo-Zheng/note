#include <hip/hip_runtime.h>

#include <chrono>
#include <iostream>
#include <memory>
#include <random>

__global__ void vectorAdd(const float* A, const float* B, float* C,
                          int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements) {
        C[i] = A[i] + B[i] + 0.0f;
    }
}

template <typename T> class CudaMemory {
public:
    explicit CudaMemory(std::size_t num)
    {
        auto err = hipMalloc((void**)&p_, num * sizeof(T));
        if (err != hipSuccess) {
            std::cerr << "Failed to allocate device: "
                      << hipGetErrorString(err) << std::endl;
            std::exit(EXIT_FAILURE);
        }
    }

    ~CudaMemory()
    {
        auto err = hipFree(p_);

        if (err != hipSuccess) {
            std::cerr << "Failed to free device: " << hipGetErrorString(err)
                      << std::endl;
            std::exit(EXIT_FAILURE);
        }
    }

    T* get() { return p_; }

private:
    T* p_ {};
};

template <typename T>
void memcpy_gpu(T* dst, T* src, std::size_t size, enum hipMemcpyKind kind)
{
    auto err = hipMemcpy(dst, src, size, kind);
    if (err != hipSuccess) {
        std::cerr << "Failed to memcpy device: " << hipGetErrorString(err)
                  << std::endl;
        std::exit(EXIT_FAILURE);
    }
}

int main()
{
    hipError_t err = hipSuccess;

    int num = 50000;
    size_t size = num * sizeof(float);
    std::cout << "num: " << num << std::endl;

    auto h_A = std::unique_ptr<float[]>(new float[num]);
    auto h_B = std::unique_ptr<float[]>(new float[num]);
    auto h_C = std::unique_ptr<float[]>(new float[num]);

    std::default_random_engine generator(
        std::chrono::system_clock::now().time_since_epoch().count());
    std::uniform_int_distribution<int> distribution(1, 100);

    for (int i = 0; i < num; ++i) {
        h_A[i] = float(distribution(generator)) / (float)100;
        h_B[i] = float(distribution(generator)) / (float)100;
    }

    // Allocate the device input vector A
    CudaMemory<float> d_A { static_cast<size_t>(num) };
    CudaMemory<float> d_B { static_cast<size_t>(num) };
    CudaMemory<float> d_C { static_cast<size_t>(num) };

    std::cout << "Copy input data from the host memory to the CUDA device"
              << std::endl;
    memcpy_gpu(d_A.get(), h_A.get(), size, hipMemcpyHostToDevice);
    memcpy_gpu(d_B.get(), h_B.get(), size, hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (num + threadsPerBlock - 1) / threadsPerBlock;
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid,
           threadsPerBlock);
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A.get(), d_B.get(),
                                                  d_C.get(), num);
    err = hipGetLastError();

    if (err != hipSuccess) {
        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    memcpy_gpu(h_C.get(), d_C.get(), size, hipMemcpyDeviceToHost);

    for (int i = 0; i < num; ++i) {
        if (fabs(h_A[i] + h_B[i] - h_C[i]) > 1e-5) {
            fprintf(stderr, "Result verification failed at element %d!\n", i);
            exit(EXIT_FAILURE);
        }
    }

    std::cout << "done" << std::endl;

    return 0;
}